#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <getopt.h>

void printPlate (double* P, int width)
{
    for (int i = 0; i < width; i++)
    {
        for (int j = 0; j < width; j++)
        {
            printf("%.1f ", P[width * j + i]);
        }
        printf("\n");
    }
}

int main (int argc, char *argv[])
{
    int n, iter, opt;
    while ((opt = getopt(argc, argv, "n:I:")) != -1)
    {
        switch (opt)
        {
            case 'I':
                iter = atoi(optarg);
            case 'n':
                n = atoi(optarg);
                break;
            default:
                exit(EXIT_FAILURE);
        }
    }
    int width = n + 2;
    int size = width * width * sizeof(double);
    double *G, *H;
    hipMallocManaged(&G, size);
    hipMallocManaged(&H, size);
    for (int i = 0; i < width; i++)
    {
        for (int j = 0; j < width; j++)
        {
            if (i == 0 || j == 0 || i == width - 1 || j == width - 1)
            {
                if (j == 0 && i > 0.3 * width && i < 0.7 * width)
                {
                    G[width * j + i] = 100.0;
                    H[width * j + i] = 100.0;
                }
                else
                {
                    G[width * j + i] = 20.0;
                    H[width * j + i] = 20.0;
                }
            }
            else
            {
                G[width * j + i] = 0.0;
                H[width * j + i] = 0.0;
            }
        }
    }
    printPlate(G, width);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop);
    int threads = sqrt(prop.maxThreadPerBlock);
    dim3 dimBlock(threads, threads);
    dim3 dimGrid(n/threads + 1, n/threads + 1);
    for (int i = 0; i < iter; i++)
    {
        hipEvent_t stop;
        hipEventCreate(&stop);
        kernel<<<dimGrid, dimBlock>>>(G, H, n);
        kernel<<<dimGrid, dimBlock>>>(H, G, n);
        hipEventSynchronize(stop);
    }
    printPlate(G, width);
    return 0;
}

__global__ void kernel(double *G, double *H, int n, int width)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    printf("%f\n ", H[width * y + x]);
    if (x < n && y < n)
    {
        G[y * width + x] = 0.25 * (H[y * width + x + 1] + H[y * width + x - 1] + H[(y - 1) * width + x] + H[(y + 1) * width + x]);
    }
    printf("\n");
}